
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 512

__global__ void add(int *a, int *b, int *c) {
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void random_ints(int* x, int size)
{
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%100;
	}
}

int main(void) {
  int *a, *b, *c; // host copies of a, b, c
  int *d_a, *d_b, *d_c; // device copies of a, b, c
  int size = N * sizeof(int);// Alloc space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  // Alloc space for host copies of a, b, c and setup input values
  a = (int *)malloc(size); random_ints(a, N);
  b = (int *)malloc(size); random_ints(b, N);
  c = (int *)malloc(size);
  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  // Launch add() kernel on GPU with N blocks
  add<<<1,N>>>(d_a, d_b, d_c);
  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  // Cleanup
  for (int i=0;i<N;i++) {
    printf("a[%d]=%d , b[%d]=%d, c[%d]=%d\n",i,a[i],i,b[i],i,c[i]);
  }

  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}
